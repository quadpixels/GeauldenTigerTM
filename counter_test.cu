#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pstm.h"

#include ""

#define USE_PSTM

#ifdef USE_PSTM
  #define SET_TXN_STATE(s) { atomicExch((int*)(&g_txnstate[tid]), (int)s); }
  #define INCREMENT_ABORT_COUNT { if (g_txnstate[tid] == ABORTED) { atomicAdd(g_n_aborts, 1); } }
  #define TX_READ(addr, ptr)      { if (!TxRead(tid, &aborted, (int*)(addr), (int*)(ptr), p_rwlog))       goto retry; }
  #define TX_READLONG(addr, ptr)  { if (!TxReadLong(tid, &aborted, (long*)(addr), (long*)(ptr), p_rwlog)) goto retry; }
  #define TX_WRITE(addr, val)     { if (!TxWrite(tid, &aborted, (int*)(addr), (int)(val), p_rwlog))       goto retry; }
  #define TX_WRITELONG(addr, val) { if (!TxWriteLong(tid, &aborted, (long*)(addr), (long)(val), p_rwlog)) goto retry; }
  #define TX_COMMIT { TxCommit(tid, &aborted, p_rwlog); }
#endif

// Note: Enable "-rdc" in Visual Studio for functions in other translation units to work
// Project Properties --> CUDA C/C++ --> Common --> Generate Relocatable Device Code
extern __device__ enum TxnState* g_txnstate;
extern __device__ int* g_n_commits, * g_n_aborts;
extern __device__ bool TxRead(const int tid, bool* aborted,	const int* const addr, int* const value, class RWLogs* rwlog);
extern __device__ bool TxWrite(const int tid, bool* aborted,	int* const addr, const int value, class RWLogs* rwlog);
extern __device__ bool TxCommit(const int tid, bool* aborted, class RWLogs* rwlog);
extern __device__ int GetThdID();

__global__ void Hello() {
  printf("Hello\n");
  __syncthreads();
}


__global__ void counterTest(class RWLogs* rwlogs, int* scratch) {
  __syncthreads();
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }

  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  bool aborted = false;
  int c;
  //TX_READ(scratch, &c);
  { if (!TxRead(tid, &aborted, (int*)scratch, (int*)&c, p_rwlog)) goto retry; }
  TX_WRITE(scratch, c + 1);
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);
}

__global__ void counterTestLong(class RWLogs* rwlogs, long* scratch) {
  __syncthreads();
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }

  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  bool aborted = false;
  int c;
  TX_READLONG(scratch, &c);
  TX_WRITELONG(scratch, c + 1);
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);
}

__global__ void counterTestMultiple(class RWLogs* rwlogs, int* scratch, const int N) {
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }
  bool aborted = false;

  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  for (int n=0; n<N; n++) {
    const int idx = (n + tid) % N;
    int c;
    TX_READ(&(scratch[idx]), &c);
    TX_WRITE(&(scratch[idx]), c+1);
  }
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);
}