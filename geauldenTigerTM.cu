#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <memory>
#include <string>
#include <assert.h>

#include ""
#include "linkedlist.h"

// Caution: sizeof(long) may be 4

// "USE_PSTM" is defined in project settings
#define USE_PSTM

#ifdef USE_PSTM
  #include "pstm.h"
  #define SET_TXN_STATE(s) { atomicExch((int*)(&g_txnstate[tid]), (int)s); }
  #define INCREMENT_ABORT_COUNT { if (g_txnstate[tid] == ABORTED) { atomicAdd(g_n_aborts, 1); } }
  #define TX_READ(addr, ptr)      { if (!TxRead(tid, &aborted, (int*)(addr), (int*)(ptr), p_rwlog))       goto retry; }
  #define TX_READLONG(addr, ptr)  { if (!TxReadLong(tid, &aborted, (long*)(addr), (long*)(ptr), p_rwlog)) goto retry; }
  #define TX_WRITE(addr, val)     { if (!TxWrite(tid, &aborted, (int*)(addr), (int)(val), p_rwlog))       goto retry; }
  #define TX_WRITELONG(addr, val) { if (!TxWriteLong(tid, &aborted, (long*)(addr), (long)(val), p_rwlog)) goto retry; }
  #define TX_COMMIT { TxCommit(tid, &aborted, p_rwlog); }
  // Book-keeping stuff for PSTM
  extern __device__           int* g_se; // SE means Shadow Entry
  extern __device__ enum TxnState* g_txnstate;
  extern __device__           int* g_locks;
  extern __device__ int g_num_blk, g_num_thd_per_blk;
#endif

#define CE(call) {\
	call; \
	hipError_t err = hipGetLastError(); \
	if(err != hipSuccess) { \
		printf("%s\n", hipGetErrorString(err)); \
		assert(0); \
	} \
}

__device__ int* g_n_commits, * g_n_aborts;

// Workloads
extern __global__ void Hello(); // This is okay; extern __device__ is not okay
extern __global__ void counterTest(class RWLogs*, int*);
extern __global__ void counterTestLong(class RWLogs* rwlogs, int64_t* scratch);
extern __global__ void counterTestMultiple(class RWLogs* rwlogs, int* scratch, const int N);
extern __global__ void counterTestMultipleLong(class RWLogs* rwlogs, int64_t* scratch, const int N);

extern __global__ void listbmk_GPU_serial(ListNode* list_head, ListNode* new_nodes, int count);

__device__ int GetThdID() {
  return threadIdx.x + blockIdx.x * blockDim.x;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

  int NB = 4, NT = 32;
  // Parameters for various tests
  int LIST_SIZE = 100;

  int run_mode = 2;
  for (int i=1; i<argc; i++) {
    int x;
    if (1 == sscanf(argv[i], "exp=%d", &x)) {
      printf("Run mode set to %d\n", x);
      run_mode = x;
    } else if (2 == sscanf(argv[i], "dim=%d,%d", &NB, &NT)) {
      printf("Dimension set to <<<%d, %d>>>\n", NB, NT);
    }
  }

  // Metadata for Orec-based STM algorithms
  #if defined(USE_PSTM)
  int* d_se, * d_locks;
  // Shadow entry
  CE(hipMalloc(&d_se, sizeof(int) * SIZE));
  CE(hipMemset(d_se, 0xFF, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_se), &d_se, sizeof(int*)));

  // Shadow entry lock
  CE(hipMalloc(&d_locks, sizeof(int) * SIZE));
  CE(hipMemset(d_locks, 0x00, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_locks), &d_locks, sizeof(int*)));
#if defined(USE_ESTM_UNDOLOG) || defined(USE_ESTM) || defined(USE_ESTM_COALESCED)
  int* d_readers;
  CE(hipMalloc(&d_readers, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemset(d_readers, 0xFF, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_readers), &d_readers, sizeof(int*)));
#endif
  enum TxnState* d_txnstate;
  CE(hipMalloc(&d_txnstate, sizeof(enum TxnState) * MAX_CONCURRENT_TXN));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_txnstate), &d_txnstate, sizeof(enum TxnState*)));

  // Read-write logs
  class RWLogs* h_rwlogs = new RWLogs[NB * NT];
  class RWLogs* d_rwlogs;
  CE(hipMalloc(&d_rwlogs, sizeof(class RWLogs) * NB * NT));
  CE(hipMemcpy(d_rwlogs, h_rwlogs, sizeof(RWLogs) * NB * NT, hipMemcpyHostToDevice));

  // Commit and abort count
  int* d_n_commits, *d_n_aborts;
  CE(hipMalloc(&d_n_commits, sizeof(int)));
  CE(hipMalloc(&d_n_aborts, sizeof(int)));
  CE(hipMemset(d_n_commits, 0x0, sizeof(int)));
  CE(hipMemset(d_n_aborts, 0x0, sizeof(int)));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_commits), &d_n_commits, sizeof(int*), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_aborts), &d_n_aborts, sizeof(int*), 0, hipMemcpyHostToDevice));

  #endif

  Hello<<<1, 1>>>();

  const int NUM_COUNTERS = 10;

  switch (run_mode) {
    case 10: { // Counte test, single int counter
      int* d_scratch, h_scratch;
      CE(hipMalloc(&d_scratch, sizeof(int)));
      CE(hipMemset(d_scratch, 0x00, sizeof(int)));
      counterTest<<<NB, NT>>>(d_rwlogs, d_scratch);
      CE(hipMemcpy(&h_scratch, d_scratch, sizeof(int), hipMemcpyDeviceToHost));
      printf("(int) Counter=%d\n", h_scratch);
      break;
    }
    case 11: { // Counter test, single long counter
      int64_t* d_scratch, h_scratch;
      CE(hipMalloc(&d_scratch, sizeof(int64_t)));
      CE(hipMemset(d_scratch, 0x00, sizeof(int64_t)));
      counterTestLong<<<NB, NT>>>(d_rwlogs, d_scratch);
      CE(hipMemcpy(&h_scratch, d_scratch, sizeof(int64_t), hipMemcpyDeviceToHost));
      printf("(long) Counter=%ld\n", h_scratch);
      break;
    }
    case 12: { // Counter test, multiple int counters
      int* d_scratch, h_scratch[NUM_COUNTERS];
      CE(hipMalloc(&d_scratch, sizeof(int)*NUM_COUNTERS));
      CE(hipMemset(d_scratch, 0x00, sizeof(int)*NUM_COUNTERS));
      counterTestMultiple<<<NB, NT>>>(d_rwlogs, d_scratch, NUM_COUNTERS);
      CE(hipMemcpy(h_scratch, d_scratch, sizeof(int)*NUM_COUNTERS, hipMemcpyDeviceToHost));
      printf("(multiple int's)");
      for (int i=0; i<NUM_COUNTERS; i++) {
        printf(" %d", h_scratch[i]);
      }
      printf("\n");
      break;
    }
    case 13: { // Counter test, multiple long counters
      int64_t* d_scratch, h_scratch[NUM_COUNTERS];
      CE(hipMalloc(&d_scratch, sizeof(int64_t)*NUM_COUNTERS));
      CE(hipMemset(d_scratch, 0x00, sizeof(int64_t)*NUM_COUNTERS));
      counterTestMultipleLong<<<NB, NT>>>(d_rwlogs, d_scratch, NUM_COUNTERS);
      CE(hipMemcpy(h_scratch, d_scratch, sizeof(int64_t)*NUM_COUNTERS, hipMemcpyDeviceToHost));
      printf("(multiple long's)");
      for (int i=0; i<NUM_COUNTERS; i++) {
        printf(" %ld", h_scratch[i]);
      }
      printf("\n");
      break;
    }
    case 20: {
      ListNode* h_listnode = new ListNode[LIST_SIZE+2];
      ListNode* d_listnode = nullptr;
      int* vals = new int[LIST_SIZE];
      for (int i=0; i<LIST_SIZE; i++) { vals[i] = i; }
      for (int i=0; i<LIST_SIZE; i++) {
        int j = rand() % (LIST_SIZE - i);
        const int tmp = vals[i];
        vals[i] = vals[j];
        vals[j] = tmp;
      }
      // Sentinel elements of the linked list
      h_listnode[0].val           = -2147483648;
      h_listnode[0].next_idx      = LIST_SIZE + 1;
      h_listnode[LIST_SIZE+1].val      = 2147483647;
      h_listnode[LIST_SIZE+1].next_idx = -999;
      for (int i=0; i<LIST_SIZE; i++) {
        h_listnode[i+1].val = vals[i];
      }
      const size_t S = sizeof(ListNode)*(2+LIST_SIZE);
      CE(hipMalloc(&d_listnode, S));
      CE(hipMemcpy(d_listnode, h_listnode, S, hipMemcpyHostToDevice));
      
      listbmk_GPU_serial<<<1, 1>>>(d_listnode, d_listnode+1, LIST_SIZE);

      CE(hipMemcpy(h_listnode, d_listnode, S, hipMemcpyDeviceToHost));
      int idx = 0;
      while (idx != -999) {
        ListNode* n = &(h_listnode[idx]);
        printf("%d ", n->val);
        idx = n->next_idx;
      }
      printf("\n");
    }
  }

  // Print statistics
  int h_n_commits, h_n_aborts;
  CE(hipMemcpy(&h_n_commits, d_n_commits, sizeof(int), hipMemcpyDeviceToHost));
  CE(hipMemcpy(&h_n_aborts, d_n_aborts, sizeof(int), hipMemcpyDeviceToHost));
  printf("%d commits, %d aborts\n", h_n_commits, h_n_aborts);

  exit(EXIT_SUCCESS);
}
