#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <memory>
#include <string>
#include <assert.h>

#include ""

// "USE_PSTM" is defined in project settings
#define USE_PSTM

#ifdef USE_PSTM
  #include "pstm.cu" // This file should be set to "not involved in the compilation process"
  #define SET_TXN_STATE(s) { atomicExch((int*)(&g_txnstate[tid]), (int)s); }
  #define INCREMENT_ABORT_COUNT { if (g_txnstate[tid] == ABORTED) { atomicAdd(g_n_aborts, 1); } }
  #define TX_READ(addr, ptr)      { if (!TxRead(tid, &aborted, (int*)(addr), (int*)(ptr), p_rwlog))       goto retry; }
  #define TX_READLONG(addr, ptr)  { if (!TxReadLong(tid, &aborted, (long*)(addr), (long*)(ptr), p_rwlog)) goto retry; }
  #define TX_WRITE(addr, val)     { if (!TxWrite(tid, &aborted, (int*)(addr), (int)(val), p_rwlog))       goto retry; }
  #define TX_WRITELONG(addr, val) { if (!TxWriteLong(tid, &aborted, (long*)(addr), (long)(val), p_rwlog)) goto retry; }
  #define TX_COMMIT { TxCommit(tid, &aborted, p_rwlog); }
#endif

#define CE(call) {\
	call; \
	hipError_t err = hipGetLastError(); \
	if(err != hipSuccess) { \
		printf("%s\n", hipGetErrorString(err)); \
		assert(0); \
	} \
}

__device__ int GetThdID() {
  return threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void counterTest(class RWLogs* rwlogs, int* scratch) {
  __syncthreads();
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }
  
  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  bool aborted = false;
  int c;
  TX_READ(scratch, &c);
  TX_WRITE(scratch, c + 1);
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);
}

__global__ void counterTestLong(class RWLogs* rwlogs, long* scratch) {
  __syncthreads();
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }

  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  bool aborted = false;
  int c;
  TX_READLONG(scratch, &c);
  TX_WRITELONG(scratch, c + 1);
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

  int NB = 4, NT = 32;

  // Metadata for Orec-based STM algorithms
  #if defined(USE_PSTM)
  int* d_se, * d_locks;
  // Shadow entry
  CE(hipMalloc(&d_se, sizeof(int) * SIZE));
  CE(hipMemset(d_se, 0xFF, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_se), &d_se, sizeof(int*)));

  // Shadow entry lock
  CE(hipMalloc(&d_locks, sizeof(int) * SIZE));
  CE(hipMemset(d_locks, 0x00, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_locks), &d_locks, sizeof(int*)));
#if defined(USE_ESTM_UNDOLOG) || defined(USE_ESTM) || defined(USE_ESTM_COALESCED)
  int* d_readers;
  CE(hipMalloc(&d_readers, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemset(d_readers, 0xFF, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_readers), &d_readers, sizeof(int*)));
#endif
  enum TxnState* d_txnstate;
  CE(hipMalloc(&d_txnstate, sizeof(enum TxnState) * MAX_CONCURRENT_TXN));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_txnstate), &d_txnstate, sizeof(enum TxnState*)));

  // Read-write logs
  class RWLogs* h_rwlogs = new RWLogs[NB * NT];
  class RWLogs* d_rwlogs;
  CE(hipMalloc(&d_rwlogs, sizeof(class RWLogs) * NB * NT));
  CE(hipMemcpy(d_rwlogs, h_rwlogs, sizeof(RWLogs) * NB * NT, hipMemcpyHostToDevice));

  // Commit and abort count
  int* d_n_commits, *d_n_aborts;
  CE(hipMalloc(&d_n_commits, sizeof(int)));
  CE(hipMalloc(&d_n_aborts, sizeof(int)));
  CE(hipMemset(d_n_commits, 0x0, sizeof(int)));
  CE(hipMemset(d_n_aborts, 0x0, sizeof(int)));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_commits), &d_n_commits, sizeof(int*), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_aborts), &d_n_aborts, sizeof(int*), 0, hipMemcpyHostToDevice));

  #endif

  int run_mode = 0;
  for (int i=1; i<argc; i++) {
    int x;
    if (1 == sscanf(argv[1], "exp=%d", &x)) {
      printf("Run mode set to %d\n", x);
      run_mode = x;
    }
  }

  switch (run_mode) {
    case 0: {
      int* d_scratch, h_scratch;
      CE(hipMalloc(&d_scratch, sizeof(int)));
      CE(hipMemset(d_scratch, 0x00, sizeof(int)));
      counterTest<<<NB, NT>>>(d_rwlogs, d_scratch);
      CE(hipMemcpy(&h_scratch, d_scratch, sizeof(int), hipMemcpyDeviceToHost));
      printf("(int) Counter=%d\n", h_scratch);
      break;
    }
    case 1: {
      long* d_scratch, h_scratch;
      CE(hipMalloc(&d_scratch, sizeof(long)));
      CE(hipMemset(d_scratch, 0x00, sizeof(long)));
      counterTestLong<<<NB, NT>>>(d_rwlogs, d_scratch);
      CE(hipMemcpy(&h_scratch, d_scratch, sizeof(long), hipMemcpyDeviceToHost));
      printf("(long) Counter=%ld\n", h_scratch);
      break;
    }
  }

  // Print statistics
  int h_n_commits, h_n_aborts;
  CE(hipMemcpy(&h_n_commits, d_n_commits, sizeof(int), hipMemcpyDeviceToHost));
  CE(hipMemcpy(&h_n_aborts, d_n_aborts, sizeof(int), hipMemcpyDeviceToHost));
  printf("%d commits, %d aborts\n", h_n_commits, h_n_aborts);

  exit(EXIT_SUCCESS);
}
