#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <memory>
#include <string>
#include <assert.h>

#include ""

#define USE_PSTM

#ifdef USE_PSTM
  #include "pstm.cu" // This file should be set to "not involved in the compilation process"
  #define SET_TXN_STATE(s) { atomicExch((int*)(&g_txnstate[tid]), (int)s); }
  #define INCREMENT_ABORT_COUNT { if (g_txnstate[tid] == ABORTED) { atomicAdd(g_n_aborts, 1); } }
  #define TX_READ(addr, ptr)      { if (!TxRead(tid, &aborted, (int*)(addr), (int*)(ptr), p_rwlog))       goto retry; }
  #define TX_READLONG(addr, ptr)  { if (!TxReadLong(tid, &aborted, (long*)(addr), (long*)(ptr), p_rwlog)) goto retry; }
  #define TX_WRITE(addr, val)     { if (!TxWrite(tid, &aborted, (int*)(addr), (int)(val), p_rwlog))       goto retry; }
  #define TX_WRITELONG(addr, val) { if (!TxWriteLong(tid, &aborted, (long*)(addr), (long)(val), p_rwlog)) goto retry; }
  #define TX_COMMIT { TxCommit(tid, &aborted, p_rwlog); }
#endif

#define CE(call) {\
	call; \
	hipError_t err = hipGetLastError(); \
	if(err != hipSuccess) { \
		printf("%s\n", hipGetErrorString(err)); \
		assert(0); \
	} \
}

__global__ void hello(int* x) {
  printf("Hello from thd (%d,%d), g_se=%p\n", blockIdx.x, threadIdx.x, g_se);
  *x = 10;
}

__device__ int GetThdID() {
  return threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void counterTest(class RWLogs* rwlogs, int* scratch) {
  __syncthreads();
  const int tid = GetThdID();
  RWLogs* p_rwlog = &(rwlogs[tid]);
  SET_TXN_STATE(RUNNING);
  p_rwlog->init();
  __threadfence();
  int attempt = 0;
  const int ATTEMPT_LIMIT = 1000000;
  printf("Thd %d start\n", tid);
retry:
  p_rwlog->releaseAll(tid);
  p_rwlog->init();
  if (attempt++ >= ATTEMPT_LIMIT) { return; }
  
  INCREMENT_ABORT_COUNT;
  SET_TXN_STATE(RUNNING);
  bool aborted = false;
  int c;
  TX_READ(scratch, &c);
  TX_WRITE(scratch, c + 1);
  TX_COMMIT;
  if (aborted) goto retry;
  else SET_TXN_STATE(ABORTED);

  printf("Tx %d counter %d->%d\n", tid, c, c + 1);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

  int NB = 4, NT = 32;

  // Metadata for Orec-based STM algorithms
  #if defined(USE_PSTM)
  int* d_se, * d_locks;
  // Shadow entry
  CE(hipMalloc(&d_se, sizeof(int) * SIZE));
  CE(hipMemset(d_se, 0xFF, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_se), &d_se, sizeof(int*)));

  // Shadow entry lock
  CE(hipMalloc(&d_locks, sizeof(int) * SIZE));
  CE(hipMemset(d_locks, 0x00, sizeof(int) * SIZE));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_locks), &d_locks, sizeof(int*)));
#if defined(USE_ESTM_UNDOLOG) || defined(USE_ESTM) || defined(USE_ESTM_COALESCED)
  int* d_readers;
  CE(hipMalloc(&d_readers, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemset(d_readers, 0xFF, sizeof(int) * SIZE * MAX_READERS_PER_ADDR));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_readers), &d_readers, sizeof(int*)));
#endif
  enum TxnState* d_txnstate;
  CE(hipMalloc(&d_txnstate, sizeof(enum TxnState) * MAX_CONCURRENT_TXN));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_txnstate), &d_txnstate, sizeof(enum TxnState*)));

  // Read-write logs
  class RWLogs* h_rwlogs = new RWLogs[NB * NT];
  class RWLogs* d_rwlogs;
  CE(hipMalloc(&d_rwlogs, sizeof(class RWLogs) * NB * NT));
  CE(hipMemcpy(d_rwlogs, h_rwlogs, sizeof(RWLogs) * NB * NT, hipMemcpyHostToDevice));

  // Commit and abort count
  int* d_n_commits, *d_n_aborts;
  CE(hipMalloc(&d_n_commits, sizeof(int)));
  CE(hipMalloc(&d_n_aborts, sizeof(int)));
  CE(hipMemset(d_n_commits, 0x0, sizeof(int)));
  CE(hipMemset(d_n_aborts, 0x0, sizeof(int)));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_commits), &d_n_commits, sizeof(int*), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol(HIP_SYMBOL(g_n_aborts), &d_n_aborts, sizeof(int*), 0, hipMemcpyHostToDevice));

  #endif

  int* d_scratch;
  CE(hipMalloc(&d_scratch, sizeof(int)));
  CE(hipMemset(d_scratch, 0x00, sizeof(int)));
  counterTest<<<NB, NT>>>(d_rwlogs, d_scratch);



  // Print statistics
  int h_n_commits, h_n_aborts;
  CE(hipMemcpy(&h_n_commits, d_n_commits, sizeof(int), hipMemcpyDeviceToHost));
  CE(hipMemcpy(&h_n_aborts, d_n_aborts, sizeof(int), hipMemcpyDeviceToHost));
  printf("%d commits, %d aborts\n", h_n_commits, h_n_aborts);

  exit(EXIT_SUCCESS);
}
